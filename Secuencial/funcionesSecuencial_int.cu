#include "hip/hip_runtime.h"
#include "funcionesSecuencial_int.h"

double GOLint(int tamanio, int iteraciones)
{

	int *malla, *aux;
	int N = tamanio;
	clock_t  inicio, final;

	malla = (int *)malloc(N * N * sizeof(int *));
	aux = (int *)malloc(N * N * sizeof(int *));
	/*
	 * Repite el proceso el mismo número de veces.
	 *  
 	*/
	leeMint(malla, aux,tamanio);
	//imprimeMint(malla, N);
	//AQUI INICIAMOS A CONTAR EL TIEMPO
	inicio = clock();
	for (int i = 0; i < iteraciones; ++i)
	{
		actualizaint(malla, aux, N);
		//printf("Iteracion %d\n", i + 1);
		//imprimeMint(malla, N);
	}
	final = clock();
	double tiempo = ((double)final - inicio) / CLOCKS_PER_SEC;
	//printf("Tiempo transcurrido con variable INT: %f [s]\n", tiempo);
	/*
 * Libera la memoria y cierra el canal con el archivo. 
 */
	free(malla);
	free(aux);
	return tiempo;
}

void leeMint(int *malla, int *aux, int tamanio)
{
	FILE *archivo = fopen(DIR_FILE, "r");
	if (archivo == NULL)
	{
		fputs("File error", stderr);
		exit(1);
	}

	char caracterAuxiliar;
	for (int i = 0; i < tamanio; i++)
	{
		for (int j = 0; j < tamanio; j++)
		{
			caracterAuxiliar = fgetc(archivo);
			if (caracterAuxiliar == '1')
			{
				aux[i * tamanio + j] = malla[i * tamanio + j] = 1;
			}
			else if (caracterAuxiliar == '0')
			{
				aux[i * tamanio + j] = malla[i * tamanio + j] = 0;
			}
		}
	}
	fclose(archivo);
}

/*
 * Funcion para imprimir la matriz 
 * Como entradas son la malla y el tamaño
 */
void imprimeMint(int *m, int N)
{
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			if (m[i * N + j] == 1)
			{
				printf("* ");
			}
			else
			{
				printf("- ");
			}
		}
		printf("\n");
	}
}

/*
 * Funcion para actualizar la matriz, intercambiandola entre la auxiliar
 * y la matriz principal.  
 */
void actualizaint(int *malla, int *aux, int N)
{
	int contador = 0;
	int celActual;
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			celActual = i * N + j;

			//Izquierda Arriba
			if (i > 0 && j > 0 && malla[celActual - N - 1] == 1)
			{
				contador++;
			}
			//Arriba
			if (i > 0 && malla[celActual - N] == 1)
			{
				contador++;
			}
			//Arriba derecha
			if (i > 0 && j < N - 1 && malla[celActual + 1 - N] == 1)
			{
				contador++;
			}
			//Izquierda
			if (j > 0 && malla[celActual - 1] == 1)
			{
				contador++;
			}
			//Derecha
			if (j < N - 1 && malla[celActual + 1] == 1)
			{
				contador++;
			}
			//Abajo izquierda
			if (i < N - 1 && j > 0 && malla[celActual + N - 1] == 1)
			{
				contador++;
			}
			//Abajo
			if (i < N - 1 && malla[celActual + N] == 1)
			{
				contador++;
			}
			//Abajo derecha
			if (i < N - 1 && j < N - 1 && malla[celActual + 1 + N] == 1)
			{
				contador++;
			}

			if (malla[celActual] == 1)
			{ //Actuamos sobre las celulas en la copia de la matriz
				if (contador == 2 || contador == 3)
				{ //La celulas vivas con 2 o 3 celulas vivas pegadas, se mantiene vivas.
					aux[celActual] = 1;
				}
				else
				{ //Si no se cumple la condicion, mueren.
					aux[celActual] = 0;
				}
			}
			else
			{
				if (contador == 3)
				{ //Las celulas muertas con 3 celulas vivas pegadas, resucitan.
					aux[celActual] = 1;
				}
			}
			contador = 0;
		}
	}
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			celActual = i * N + j;
			malla[celActual] = aux[celActual]; //Copiamos la matriz origen en destino
		}
	}
}
