#include "hip/hip_runtime.h"
#include "funcionesSecuencial_float.h"

double GOLfloat(int tamanio, int iteraciones)
{
	float *malla, *aux;
	int N = tamanio;
	clock_t inicio, final;

	malla = (float *)malloc(N * N * sizeof(float *));
	aux = (float *)malloc(N * N * sizeof(float *));
	/*
	 * Repite el proceso el mismo número de veces.
	 *  
 	*/
	leeMfloat(malla, aux,tamanio);
	//imprimeMfloat(malla, N);
	//AQUI INICIAMOS A CONTAR EL TIEMPO
	inicio=clock();
	for (int i = 0; i < iteraciones; ++i)
	{
		actualizafloat(malla, aux, N);
		//printf("Iteracion %d\n", i + 1);
		//imprimeMfloat(malla, N);
	}
	final = clock();
	double tiempo = ((double)final - inicio) / CLOCKS_PER_SEC;
	//printf("Tiempo transcurrido con variable FLOAT: %f [s]\n", tiempo);
	/*
 * Libera la memoria y cierra el canal con el archivo. 
 */
	free(malla);
	free(aux);
	return tiempo;
}

void leeMfloat(float *malla, float *aux,int tamanio)
{
	FILE *archivo = fopen(DIR_FILE, "r");
	if (archivo == NULL)
	{
		fputs("File error", stderr);
		exit(1);
	}

	char caracterAuxiliar;
	for (int i = 0; i < tamanio; i++)
	{
		for (int j = 0; j < tamanio; j++)
		{
			caracterAuxiliar = fgetc(archivo);
			if (caracterAuxiliar == '1')
			{
				aux[i * tamanio + j] = malla[i * tamanio + j] = 1.0;
			}
			else if (caracterAuxiliar == '0')
			{
				aux[i * tamanio + j] = malla[i * tamanio + j] = 0.0;
			}
		}
	}
	fclose(archivo);
}

/*
 * Funcion para imprimir la matriz 
 * Como entradas son la malla y el tamaño
 */
void imprimeMfloat(float *m, int N)
{
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			if (m[i * N + j] == 1)
			{
				printf("* ");
			}
			else
			{
				printf("- ");
			}
		}
		printf("\n");
	}
}

/*
 * Funcion para actualizar la matriz, intercambiandola entre la auxiliar
 * y la matriz principal.  
 */
void actualizafloat(float *malla, float *aux, int N)
{
	int contador = 0;
	int celActual;
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			celActual = i * N + j;

			//Izquierda Arriba
			if (i > 0 && j > 0 && malla[celActual - N - 1] == 1.0)
			{
				contador++;
			}
			//Arriba
			if (i > 0 && malla[celActual - N] == 1.0)
			{
				contador++;
			}
			//Arriba derecha
			if (i > 0 && j < N - 1 && malla[celActual + 1 - N] == 1.0)
			{
				contador++;
			}
			//Izquierda
			if (j > 0 && malla[celActual - 1] == 1.0)
			{
				contador++;
			}
			//Derecha
			if (j < N - 1 && malla[celActual + 1] == 1.0)
			{
				contador++;
			}
			//Abajo izquierda
			if (i < N - 1 && j > 0 && malla[celActual + N - 1] == 1.0)
			{
				contador++;
			}
			//Abajo
			if (i < N - 1 && malla[celActual + N] == 1.0)
			{
				contador++;
			}
			//Abajo derecha
			if (i < N - 1 && j < N - 1 && malla[celActual + 1 + N] == 1.0)
			{
				contador++;
			}

			if (malla[celActual] == 1.0)
			{ //Actuamos sobre las celulas en la copia de la matriz
				if (contador == 2 || contador == 3)
				{ //La celulas vivas con 2 o 3 celulas vivas pegadas, se mantiene vivas.
					aux[celActual] = 1.0;
				}
				else
				{ //Si no se cumple la condicion, mueren.
					aux[celActual] = 0.0;
				}
			}
			else
			{
				if (contador == 3)
				{ //Las celulas muertas con 3 celulas vivas pegadas, resucitan.
					aux[celActual] = 1.0;
				}
			}
			contador = 0;
		}
	}
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			celActual = i * N + j;
			malla[celActual] = aux[celActual]; //Copiamos la matriz origen en destino
		}
	}
}
