#include "hip/hip_runtime.h"
#include "funcionesParalelo_double.h"
#include <math.h>
double GOL_Pdouble(int N, int iteraciones, int MAX_threads){
    double *tablero, *tablero_aux;
	double *d_tablero, *d_tablero_aux;
	clock_t  inicio, final;
    size_t size = N*N*sizeof(double);

    //Asignacion de memoria del lado del host
    tablero = (double*)malloc(size);
    tablero_aux = (double*)malloc(size);
    

    //Asignacion de memoria del lado de device
    hipMalloc(&d_tablero, size);
    hipMalloc(&d_tablero_aux, size);
	
	FILE * archivo = fopen(DIR_FILE, "r");
	if (archivo==NULL) {fputs ("File error",stderr); exit (1);}

	char caracterAuxiliar;

	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			caracterAuxiliar = fgetc(archivo);
			if (caracterAuxiliar == '1'){
				tablero_aux[i*N+j]=tablero[i*N+j]=1.0;
			}
			else {
				tablero_aux[i*N+j]=tablero[i*N+j]=0.0;
			}
		}
	}

	fclose(archivo);


	inicio = clock(); //tiempo inicial

    hipMemcpy(d_tablero,tablero,size,hipMemcpyHostToDevice);
    hipMemcpy(d_tablero_aux,tablero_aux,size,hipMemcpyHostToDevice);

	//VOLVER CONSTANTE
	int dim_MAX = (int)sqrt(MAX_threads);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((dim_MAX + dimBlock.x-1)/dimBlock.x, (dim_MAX+dimBlock.y-1)/dimBlock.y);
	//dim3 dimGrid(2,9);

	//imprimeM_P_double<<<1,1>>>(d_tablero);
	//hipDeviceSynchronize();
	for (int i = 0; i < iteraciones; ++i)
	{
	 	//printf("Iteracion %d\n",i+1);
		actualiza_Pdouble<<<dimGrid,dimBlock>>>(d_tablero, d_tablero_aux, N, dim_MAX);
		hipDeviceSynchronize();//espera a que todos los hilos terminen su ejecución
		copiaMatriz_Pdouble<<<dimGrid,dimBlock>>>(d_tablero_aux, d_tablero, N, dim_MAX);
		hipDeviceSynchronize();
		//imprimeM_Pdouble<<<1,1>>>(d_tablero,N);
		//hipDeviceSynchronize();
	}
	final = clock();
	double tiempo = ((double)final - inicio) / CLOCKS_PER_SEC;
	//printf("el tiempo final es %f\n", tiempo);
	//timepo final

    free(tablero);
    free(tablero_aux);

    hipFree(d_tablero);
    hipFree(d_tablero_aux);

    return tiempo;
}

/*
 * Funcion para actualizar la matriz, intercambiandola entre la auxiliar
 * y la matriz principal.  
 */
 __global__ void actualiza_Pdouble(double *malla, double *aux, int N, int dim_MAX){
	int contador=0;
	int celActual;
	int i = blockDim.x * blockIdx.x +  threadIdx.x; //fila
	int j = blockDim.y * blockIdx.y + threadIdx.y; //Columna
	if (i < N && j < N) {
		int ii = (int)(N/dim_MAX)+1;
		for(int k = 0; k < ii; k++){
			celActual = i*N+j;
			if(celActual < N*N){
				//printf("%d ",celActual);
				//Izquierda Arriba
				if(i>0 && j>0 && malla[celActual-N-1]==1){
					contador++;
				}
				//Arriba
				if(i>0 && malla[celActual-N]==1){
					contador++;
				}
				//Arriba derecha
				if(i>0 && j<N-1 && malla[celActual+1-N]==1){
					contador++;
				}
				//Izquierda
				if(j>0 && malla[celActual-1]==1){
					contador++;
				}
				//Derecha
				if(j<N-1 && malla[celActual+1]==1){
					contador++;
				}
				//Abajo izquierda
				if(i<N-1 && j>0 && malla[celActual+N-1]==1){
					contador++;
				}
				//Abajo
				if(i<N-1 && malla[celActual+N]==1){
					contador++;
				}
				//Abajo derecha
				if(i<N-1 && j<N-1 && malla[celActual+1+N]==1){
					contador++;
				}
				if(malla[celActual]==1){		//Actuamos sobre las celulas en la copia de la matriz
					if(contador==2 || contador==3){//La celulas vivas con 2 o 3 celulas vivas pegadas, se mantiene vivas.
						aux[celActual]=1;
					}
					else{					//Si no se cumple la condicion, mueren.
						aux[celActual]=0;
						}
					}
				else{
					if(contador==3){		//Las celulas muertas con 3 celulas vivas pegadas, resucitan.
						aux[celActual]=1;
						}
					}
				contador=0;
				}	
			celActual = celActual + dim_MAX;
		}	
	}
}

__global__ void copiaMatriz_Pdouble(double *malla, double *aux, int N, int dim_MAX){
	int celActual;
	int i = blockDim.x * blockIdx.x +  threadIdx.x; //fila
	int j = blockDim.y * blockIdx.y + threadIdx.y; //Columna
	if (i < N && j < N) {
		int ii = (int)(N/dim_MAX)+1;
		for(int k = 0; k < ii; k++){
			celActual = i*N+j;
			if(celActual < N*N){
				aux[celActual] = malla[celActual];
				celActual = celActual + dim_MAX;
			}
		}
	}
}


/*
 * Funcion para imprimir la matriz 
 * Como entradas son la malla y el tamaño
 */
__global__ void imprimeM_Pdouble(double *m, int N){
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			if (m[i*N+j]==1)
			{
				printf("* ");
			}
			else{
				printf("- ");
			}
		}
		printf("\n");
	}
}